#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fftx3.hpp"
#include "fftx_mdprdft_public.h"
#include "device_macros.h"

#include <stdlib.h>
#include <string.h>

static int M, N, K;
static bool writefiles = false;

//  generate file name

static char * generateFileName ( const char *type )
{
  // type is:
  // input ==> random input data;
  // output ==> spiral output data;
  // roc ==> rocFFT output data
  static char fileNameBuff[100];
  sprintf ( fileNameBuff, "mdprdft3d-%s-%dx%dx%d.dat", type, M, N, K );
  return fileNameBuff;
}

//  write data to file(s) for test repeatability.

static void writeBufferToFile ( const char *type, double *datap )
{
  char *fname = generateFileName ( type );
  FILE *fhandle = fopen ( fname, "w" );
  fprintf ( fhandle, "[ \n" );
  for ( int mm = 0; mm < M; mm++ )
    {
      for ( int nn = 0; nn < N; nn++ )
        {
          for ( int kk = 0; kk < K; kk++ )
            {
              // Output is complex
              fprintf ( fhandle, "FloatString(\"%.12g\"), FloatString(\"%.12g\"), ",
                        datap[(kk + nn*K + mm*N*K)*2 + 0],
                        datap[(kk + nn*K + mm*N*K)*2 + 1] );
              if ( kk > 0 && kk % 8 == 0 )
                fprintf ( fhandle, "\n" );
            }
          fprintf ( fhandle, "\n" );
        }
    }
  fprintf ( fhandle, "];\n" );
	
  //  fwrite ( datap, sizeof(double) * 2, M * N * K, fhandle );
  fclose ( fhandle );
  return;
}

static void buildInputBuffer ( double *host_X, double *X, int genData )
{
  if ( genData )
    { // generate a new data input buffer
      for (int m = 0; m < M; m++)
        {
          for (int n = 0; n < N; n++)
            {
              for (int k = 0; k < K; k++)
                {
                  // Input is real.
                  host_X[k + n*K + m*N*K] =
                    1 - ((double) rand()) / (double) (RAND_MAX/2);
                  // host_X[(k + n*K + m*N*K)*2 + 0] =
                  //   1 - ((double) rand()) / (double) (RAND_MAX/2);
                  // host_X[(k + n*K + m*N*K)*2 + 1] =
                  //   1 - ((double) rand()) / (double) (RAND_MAX/2);
                }
            }
        }
    }

  DEVICE_MEM_COPY ( X, host_X,
                    (M * N * K * sizeof(double)), // If complex, 2 *
                    MEM_COPY_HOST_TO_DEVICE);
  DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );
  return;
}

static void checkOutputBuffers ( double *Y, double *cufft_Y )
{
  printf("cube = [ %d, %d, %d ]\t", M, N, K);
  DEVICE_FFT_DOUBLECOMPLEX *host_Y       = new DEVICE_FFT_DOUBLECOMPLEX[M*N*K];
  DEVICE_FFT_DOUBLECOMPLEX *host_cufft_Y = new DEVICE_FFT_DOUBLECOMPLEX[M*N*K];

  DEVICE_MEM_COPY ( host_Y,             Y,
                    (M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX)),
                    MEM_COPY_DEVICE_TO_HOST );
  DEVICE_MEM_COPY ( host_cufft_Y, cufft_Y,
                    (M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX)),
                    MEM_COPY_DEVICE_TO_HOST );

  bool correct = true;
  //  int errCount = 0;
  double maxdelta = 0.0;

  for ( int m = 0; m < M; m++ )
    {
      for ( int n = 0; n < N; n++ )
        {
          for ( int k = 0; k < K; k++ )
            {
              DEVICE_FFT_DOUBLECOMPLEX s = host_Y      [k + n*K + m*N*K];
              DEVICE_FFT_DOUBLECOMPLEX c = host_cufft_Y[k + n*K + m*N*K];
	    
              bool elem_correct =
                ( abs(s.x - c.x) < 1e-7 ) &&
                ( abs(s.y - c.y) < 1e-7 );
              maxdelta = maxdelta < (double)(abs(s.x -c.x)) ?
                (double)(abs(s.x -c.x)) : maxdelta ;
              maxdelta = maxdelta < (double)(abs(s.y -c.y)) ?
                (double)(abs(s.y -c.y)) : maxdelta ;

              correct &= elem_correct;
              //  if (!elem_correct && errCount < 10) {
              //  	correct = false;
              //  	errCount++;
              //      printf("error at (%d,%d,%d): %f+%fi instead of %f+%fi\n", k, n, m, s.x, s.y, c.x, c.y);
              //  }
            }
        }
    }
	
  printf ( "Correct: %s\tMax delta = %E\t\t##PICKME## \n", (correct ? "True" : "False"), maxdelta );
  fflush ( stdout );

  if ( writefiles )
    {
      writeBufferToFile ( (const char *)"spiral-out",
                          (double *)host_Y );
      writeBufferToFile ( (const char *)"rocFFT",
                          (double *)host_cufft_Y );
    }
  delete[] host_Y;
  delete[] host_cufft_Y;

  return;
}


static int NUM_ITERS = 100;

int main( int argc, char** argv)
{
  int iloop = 0;
  bool oneshot = false;
  int iters = NUM_ITERS + 10;

  //  Test is to time on a GPU [CUDA or HIP], check library support this mode
  if ( fftx_mdprdft_GetLibraryMode () != LIB_MODE_CUDA &&
       fftx_mdprdft_GetLibraryMode () != LIB_MODE_HIP )
    {
      printf ( "%s: fftx_mdprdft library doesn't support GPU, exiting...\n",
               argv[0] );
      exit (-1);
    }

  printf ( "Usage: %s: [ iterations ] [ size: MMxNNxKK ] [ writefiles ]\n",
           argv[0] );
  if ( argc > 1 )
    {
      NUM_ITERS = atoi ( argv[1] );
      iters = NUM_ITERS + 10;
      printf ( "%s: Measure %d iterations, ", argv[0], iters );
		
      if ( argc > 2 )
        {
          char * foo = argv[2];
          M = atoi ( foo );
          while ( * foo != 'x' ) foo++;
          foo++ ;
          N = atoi ( foo );
          while ( * foo != 'x' ) foo++;
          foo++ ;
          K = atoi ( foo );
          oneshot = true;
          printf ( "Run size: %dx%dx%d, ", M, N, K );

          if ( argc > 3 )
            {
              //  Only write files when a specified [single] size is used.  Write data to
              //  files -- spiral input data, spiral output data, and rocFFT/cuFFT output
              writefiles = true;
            }
          printf ( "%s data files\n", (writefiles) ? "WRITE" : "DO NOT write" );
        }
      else
        {
          printf ( "Run all sizes found in library, " );
          printf ( "%s data files\n", (writefiles) ? "WRITE" : "DO NOT write" );
        }
    }
  else
    {
      printf ( "%s: Measure %d iterations for all sizes found in the library\n",
               argv[0], iters );
    }
						  
  fftx::point_t<3> *wcube, curr;

  wcube = fftx_mdprdft_QuerySizes ();
  if (wcube == NULL)
    {
      printf ( "%s: Failed to get list of available sizes\n", argv[0] );
      exit (-1);
    }

  if ( oneshot )
    {
      for ( iloop = 0; ; iloop++ )
        {
          if ( wcube[iloop].x[0] == 0 &&
               wcube[iloop].x[1] == 0 &&
               wcube[iloop].x[2] == 0 )
            {
              //  requested size is not in library, print message & exit
              printf ( "%s: Cube { %d, %d, %d } not found in library ... exiting\n",
                       argv[0], M, N, K );
              exit (-1);
            }
          if ( wcube[iloop].x[0] == M &&
               wcube[iloop].x[1] == N &&
               wcube[iloop].x[2] == K )
            {
              break;
            }
        }
    }

#if defined(FFTX_HIP)
  //  setup the library
  rocfft_setup();
#endif
	
  DEVICE_EVENT_T start, stop, custart, custop;
  DEVICE_EVENT_CREATE ( &start );
  DEVICE_EVENT_CREATE ( &stop );
  DEVICE_EVENT_CREATE ( &custart );
  DEVICE_EVENT_CREATE ( &custop );

  double *X, *Y;
  double sym[100];  // dummy symbol
  transformTuple_t *tupl;

  for ( /* iloop is initialized */ ; ; iloop++ )
    {
      curr = wcube[iloop];
      if ( curr.x[0] == 0 &&
           curr.x[1] == 0 &&
           curr.x[2] == 0 ) break;

      printf ( "Cube size { %d, %d, %d } is available\n",
               curr.x[0], curr.x[1], curr.x[2]);
      tupl = fftx_mdprdft_Tuple ( wcube[iloop] );
      if ( tupl == NULL )
        {
          printf ( "Failed to get tuple for cube { %d, %d, %d }\n",
                   curr.x[0], curr.x[1], curr.x[2]);
        }
      else
        {
          M = curr.x[0], N = curr.x[1], K = curr.x[2];
          DEVICE_MALLOC ( &X, ( M * N * K * sizeof(double) ) ); // complex has 2 *
          DEVICE_MALLOC ( &Y, ( M * N * K * 2 * sizeof(double) ) );

          double *host_X = new double[ M * N * K ]; // complex has 2 *
          DEVICE_FFT_DOUBLECOMPLEX *cufft_Y; 
          DEVICE_MALLOC ( &cufft_Y, ( M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );

          //  want to run and time: 1st iteration; 2nd iteration; then N iterations
          //  Report 1st time, 2nd time, and average of N further iterations
          float *milliseconds   = new float[iters];
          float *cumilliseconds = new float[iters];
          bool check_buff = true;

          DEVICE_FFT_HANDLE plan;
          DEVICE_FFT_RESULT res;
          res = DEVICE_FFT_PLAN3D ( &plan, M, N, K, DEVICE_FFT_D2Z );
          if ( res != DEVICE_FFT_SUCCESS ) {
            printf ( "Create DEVICE_FFT_PLAN3D failed with error code %d ... skip buffer check\n", res );
            check_buff = false;
          }

          //  Call the transform init function
          ( * tupl->initfp )();
          DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

          // set up data in input buffer
          buildInputBuffer(host_X, X, 1);
          if ( writefiles )
            {
              printf ( "Write input buffer to a file..." );
              writeBufferToFile ( (const char *)"input", host_X );
              printf ( "done\n" );
            }

          for ( int ii = 0; ii < iters; ii++ )
            {
              //  Call the main transform function
              DEVICE_EVENT_RECORD ( start );
              ( * tupl->runfp ) ( Y, X, sym );
              DEVICE_EVENT_RECORD ( stop );
              DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

              DEVICE_EVENT_SYNCHRONIZE ( stop );
              DEVICE_EVENT_ELAPSED_TIME ( &milliseconds[ii], start, stop );

              /* #ifdef USE_DIFF_DATA */
              /* 				buildInputBuffer(host_X, X, 1); */
              /* #else */
              /* 				buildInputBuffer(host_X, X, 0); */
              /* #endif */
            }

          //  Call the destroy function
          ( * tupl->destroyfp )();
          DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

          if ( check_buff )
            {
              for ( int ii = 0; ii < iters; ii++ )
                {
                  DEVICE_EVENT_RECORD ( custart );
                  res = DEVICE_FFT_EXECD2Z ( plan,
                                             (DEVICE_FFT_DOUBLEREAL *) X,
                                             (DEVICE_FFT_DOUBLECOMPLEX *) cufft_Y );
                  // C2C has additional argument DEVICE_FFT_FORWARD
                  if ( res != DEVICE_FFT_SUCCESS)
                    {
                      printf ( "Launch DEVICE_FFT_EXECD2Z failed with error code %d ... skip buffer check\n", res );
                      check_buff = false;
                      break;
                    }
                  DEVICE_EVENT_RECORD ( custop );
                  DEVICE_EVENT_SYNCHRONIZE ( custop );
                  DEVICE_EVENT_ELAPSED_TIME ( &cumilliseconds[ii], custart, custop );

                  /* #ifdef USE_DIFF_DATA */
                  /* 				buildInputBuffer(host_X, X, 1); */
                  /* #else */
                  /* 				buildInputBuffer(host_X, X, 0); */
                  /* #endif */
                }
            }
          DEVICE_SYNCHRONIZE ();

          //  check cufft/rocfft and FFTX got same results
          if ( check_buff ) checkOutputBuffers ( Y, (double *)cufft_Y );

          //  printf("cube = [ %d, %d, %d ]\t\t ##PICKME## \n", M, N, K);
          printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tFIRST iteration\t##PICKME## \n",
                 milliseconds[0], cumilliseconds[0]);
          printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tSECOND iteration\t##PICKME## \n",
                 milliseconds[1], cumilliseconds[1]);

          float cumulSpiral = 0.0, cumulHip = 0.0;
          for ( int ii = 10; ii < iters; ii++ )
            {
              cumulSpiral += milliseconds[ii];
              cumulHip    += cumilliseconds[ii];
            } 
          printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft), AVERAGE over %d iterations (range: 11 - %d) ##PICKME## \n",
                 cumulSpiral / NUM_ITERS, cumulHip / NUM_ITERS, NUM_ITERS, (10 + NUM_ITERS) );

          DEVICE_FREE ( X );
          DEVICE_FREE ( Y );
          DEVICE_FREE ( cufft_Y );
          delete[] host_X;
          delete[] milliseconds;
          delete[] cumilliseconds;

          if ( oneshot ) break;
        }
    }

#if defined(FFTX_HIP)
  //  cleanup the library
  rocfft_cleanup();
#endif
}
