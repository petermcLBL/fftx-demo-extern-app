#include "hip/hip_runtime.h"
#include <stdio.h>

#include "device_macros.h"

#include "fftx_mddft_public.h"
#include "fftx_imddft_public.h"
#include "fftx_mdprdft_public.h"
#include "fftx_imdprdft_public.h"
// #include "fftx_rconv_public.h"

#include "fftx3utilities.h"

enum TransformType { MDDFT, IMDDFT, MDPRDFT, IMDPRDFT };

static bool writefiles = false;

template<typename T>
T avgSubarray(const T* arr, int lo, int hi)
{
  T tot = 0.;
  int len = 0;
  for (int i = lo; i <= hi; i++)
    {
      tot += arr[i];
      len++;
    }
  T avg = tot / (len * 1.);
  return avg;
}

void setRand(double& a_val)
{
  a_val = 1. - ((double) rand()) / (double) (RAND_MAX/2);
}

void setRand(std::complex<double>& a_val)
{
  double x, y;
  setRand(x);
  setRand(y);
  a_val = std::complex<double>(x, y);
}

double diffAbs(double a_x,
               double a_y)
{
  double diffNorm = a_x - a_y;
  if (diffNorm < 0.) diffNorm = -diffNorm;
  return diffNorm;
}

double diffAbs(std::complex<double>& a_x,
               std::complex<double>& a_y)
{
  double diffNorm = std::abs(a_x - a_y);
  return diffNorm;
}

DEVICE_FFT_RESULT deviceExecD2Z(DEVICE_FFT_HANDLE a_plan,
                                double* a_in,
                                std::complex<double>* a_out)
{
  return DEVICE_FFT_EXECD2Z(a_plan,
                            (DEVICE_FFT_DOUBLEREAL*) a_in,
                            (DEVICE_FFT_DOUBLECOMPLEX*) a_out);
}


template<typename T_IN, typename T_OUT>
struct deviceTransform
{
  deviceTransform(DEVICE_FFT_TYPE a_tp,
                  int a_dir = 0)
  {
    m_tp = a_tp;
    m_dir = a_dir;
  }
                  
  DEVICE_FFT_TYPE m_tp;

  int m_dir;

  DEVICE_FFT_RESULT plan3d(DEVICE_FFT_HANDLE& a_plan,
                           fftx::point_t<3> a_tfmSize)
  {
    return DEVICE_FFT_PLAN3D(&a_plan,
                             a_tfmSize[0], a_tfmSize[1], a_tfmSize[2],
                             m_tp);
  }

  DEVICE_FFT_RESULT exec(DEVICE_FFT_HANDLE a_plan,
                         T_IN* a_in,
                         T_OUT* a_out)
  {
    if (m_tp == DEVICE_FFT_Z2Z)
      {
        return DEVICE_FFT_EXECZ2Z(a_plan,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_out,
                                  m_dir);
      }
    else if (m_tp == DEVICE_FFT_D2Z)
      {
        return DEVICE_FFT_EXECD2Z(a_plan,
                                  (DEVICE_FFT_DOUBLEREAL*) a_in,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_out);
      }
    else if (m_tp == DEVICE_FFT_Z2D)
      {
        return DEVICE_FFT_EXECZ2D(a_plan,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (DEVICE_FFT_DOUBLEREAL*) a_out);
      }
    else
      {
        return (DEVICE_FFT_RESULT) -1;
      }
  }
};
  

deviceTransform<std::complex<double>, std::complex<double> >
mddftDevice(DEVICE_FFT_Z2Z, DEVICE_FFT_FORWARD);

deviceTransform<std::complex<double>, std::complex<double> >
imddftDevice(DEVICE_FFT_Z2Z, DEVICE_FFT_INVERSE);

deviceTransform<double, std::complex<double> >
mdprdftDevice(DEVICE_FFT_D2Z);

deviceTransform<std::complex<double>, double>
imdprdftDevice(DEVICE_FFT_Z2D);

template<typename T_IN, typename T_OUT>
void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                T_IN* a_inPtr,
                T_OUT* a_outPtr);

void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                std::complex<double>* a_inPtr,
                std::complex<double>* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
}


void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                double* a_inPtr,
                std::complex<double>* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
  // Halve the domain of the complex array.
#if FFTX_COMPLEX_TRUNC_LAST
  a_outSize[2] = a_outSize[2]/2 + 1;
#else
  a_outSize[0] = a_outSize[0]/2 + 1;
#endif
}


void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                std::complex<double>* a_inPtr,
                double* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
  // Halve the domain of the complex array.
#if FFTX_COMPLEX_TRUNC_LAST
  a_inSize[2] = a_inSize[2]/2 + 1;
#else
  a_inSize[0] = a_inSize[0]/2 + 1;
#endif
}


static int NUM_ITERS = 100;
static int BASE_ITERS = 10;

template<typename T_IN, typename T_OUT>
void compareSize(fftx::point_t<3> a_size,
                 transformTuple_t *a_tupl,
                 deviceTransform<T_IN, T_OUT>& a_tfmDevice)
{
  bool doDevice = true;
  bool doSpiral = true;

  if (a_tupl == NULL)
    {
      doSpiral = false;
      printf ( "Failed to get tuple for cube { %d, %d, %d }\n",
               a_size[0], a_size[1], a_size[2]);
    }

  /*
    Allocate space for arrays, and set input array.
  */
  fftx::point_t<3> inputSize, outputSize;
  T_IN* inPtr;
  T_OUT* outPtr;
  inoutSizes(inputSize, outputSize, a_size, inPtr, outPtr);
  

  // This doesn't work. :/
  // const fftx::point_t<3> unit = fftx::point_t<3>::Unit();
  //   fftx::box_t<3> inputDomain(unit, inputSize);
  //   fftx::box_t<3> outputDomain(unit, outputSize);

  fftx::box_t<3> inputDomain(fftx::point_t<3>({{1, 1, 1}}),
                             fftx::point_t<3>({{inputSize[0],
                                                inputSize[1],
                                                inputSize[2]}}));
  fftx::box_t<3> outputDomain(fftx::point_t<3>({{1, 1, 1}}),
                              fftx::point_t<3>({{outputSize[0],
                                                 outputSize[1],
                                                 outputSize[2]}}));
  
  fftx::array_t<3, T_IN> inputArrayHost(inputDomain);
  size_t nptsInput = inputDomain.size();
  size_t nptsOutput = outputDomain.size();
  size_t bytesInput = nptsInput * sizeof(T_IN);
  size_t bytesOutput = nptsOutput * sizeof(T_OUT);
  forall([](T_IN(&v), const fftx::point_t<3>& p)
         {
           setRand(v);
         }, inputArrayHost);
  // This symmetrizes only for complex input and real output,
  // in order to get a complex array that transforms to a real array.
  fftx::array_t<3, T_OUT> outputArrayHost(outputDomain);
  symmetrizeHermitian(inputArrayHost, outputArrayHost);

  T_IN* inputHostPtr = inputArrayHost.m_data.local();
  // additional code for GPU programs
  T_IN* inputDevicePtr;
  T_OUT* outputSpiralDevicePtr;
  T_OUT* outputDeviceFFTDevicePtr;
  DEVICE_MALLOC(&inputDevicePtr, bytesInput);
  DEVICE_MALLOC(&outputSpiralDevicePtr, bytesOutput);
  DEVICE_MALLOC(&outputDeviceFFTDevicePtr, bytesOutput);
  // Do this at the beginning of each iteration instead of here.
  //  DEVICE_MEM_COPY(inputDevicePtr, inputHostPtr, // dest, source
  //                  npts*sizeof(double), // bytes
  //                  MEM_COPY_HOST_TO_DEVICE); // type
  
  /*
    Set up timers for deviceFFT.
   */
  DEVICE_EVENT_T spiralFFT_start, spiralFFT_stop;
  DEVICE_EVENT_T deviceFFT_start, deviceFFT_stop;
  DEVICE_EVENT_CREATE ( &spiralFFT_start );
  DEVICE_EVENT_CREATE ( &spiralFFT_stop );
  DEVICE_EVENT_CREATE ( &deviceFFT_start );
  DEVICE_EVENT_CREATE ( &deviceFFT_stop );

  int iters = NUM_ITERS + BASE_ITERS;

  /*
    Get plan for deviceFFT.
  */
  // printf("get deviceFFT plan\n");
  DEVICE_FFT_HANDLE plan;
  {
    auto rc = a_tfmDevice.plan3d(plan, a_size);
    if (rc != DEVICE_FFT_SUCCESS)
      {
        printf ( "Create DEVICE_FFT_PLAN3D failed with error code %d ... skip buffer check\n",
                 rc );
        doDevice = false;
      }
  }

  /*
    Time iterations of real-to-complex deviceFFT calls using the plan.
   */
  // printf("call deviceExec %d times\n", a_iterations);

  float* deviceFFT_gpu = new float[iters];
  for (int i = 0; i < iters; i++)
    {
      deviceFFT_gpu[i] = 0.;
    }
  if (doDevice)
    {
      for (int itn = 0; itn < iters; itn++ )
        {
          DEVICE_MEM_COPY(inputDevicePtr, // dest
                          inputHostPtr, // source
                          bytesInput, // bytes
                          MEM_COPY_HOST_TO_DEVICE); // type
          DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR() );
          DEVICE_EVENT_RECORD( deviceFFT_start );
          int rc = a_tfmDevice.exec(plan,
                                    inputDevicePtr,
                                    outputDeviceFFTDevicePtr);
          if (rc != DEVICE_FFT_SUCCESS)
            {
              printf ( "Launch device exec failed with error code %d ... skip buffer check\n",
                       rc );
              doDevice = false;
              break;
            }
          DEVICE_EVENT_RECORD( deviceFFT_stop );
          DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR() );
          DEVICE_EVENT_SYNCHRONIZE( deviceFFT_stop );
          DEVICE_EVENT_ELAPSED_TIME( &deviceFFT_gpu[itn],
                                     deviceFFT_start,
                                     deviceFFT_stop );
        }
    }
  DEVICE_FFT_DESTROY(plan);

  DEVICE_SYNCHRONIZE();

  // printf("call Spiral transform %d times\n", iters);

  /*
    Time iterations of transform with SPIRAL-generated code.
   */
  float* spiral_gpu = new float[iters];
  for (int i = 0; i < iters; i++)
    {
      spiral_gpu[i] = 0.;
    }

  DEVICE_MEM_COPY(inputDevicePtr, // dest
                  inputHostPtr, // source
                  bytesInput, // bytes
                  MEM_COPY_HOST_TO_DEVICE); // type
  DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR() );

  if (doSpiral)
    {
      double sym[100];  // dummy symbol
      ( * a_tupl->initfp )();
      DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

      for (int itn = 0; itn < iters; itn++)
        {
          DEVICE_EVENT_RECORD( spiralFFT_start );
          ( * a_tupl->runfp ) ( (double*) outputSpiralDevicePtr,
                                (double*) inputDevicePtr,
                                sym );
          DEVICE_EVENT_RECORD( spiralFFT_stop );
          DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );
          DEVICE_EVENT_SYNCHRONIZE ( spiralFFT_stop );
          DEVICE_EVENT_ELAPSED_TIME ( &spiral_gpu[itn],
                                      spiralFFT_start,
                                      spiralFFT_stop );
        }

      //  Call the destroy function
      ( * a_tupl->destroyfp )();
      DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );
    }
  
  /*
    Check that deviceFFT and SPIRAL give the same results on last iteration.
  */
  T_OUT* outputSpiralHostPtr = new T_OUT[nptsOutput];
  T_OUT* outputDeviceFFTHostPtr = new T_OUT[nptsOutput];
  DEVICE_MEM_COPY(outputSpiralHostPtr, // dest
                  outputSpiralDevicePtr, // source
                  bytesOutput, // bytes
                  MEM_COPY_DEVICE_TO_HOST); // type
  DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR() );
  DEVICE_MEM_COPY(outputDeviceFFTHostPtr, // dest
                  outputDeviceFFTDevicePtr, // source
                  bytesOutput, // bytes
                  MEM_COPY_DEVICE_TO_HOST); // type
  DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR() );

  DEVICE_FREE(inputDevicePtr);
  DEVICE_FREE(outputSpiralDevicePtr);
  DEVICE_FREE(outputDeviceFFTDevicePtr);

  printf("cube = [ %d, %d, %d ]\t", a_size[0], a_size[1], a_size[2]);
  if (doSpiral && doDevice)
    {
      bool correct = true;
      const double tol = 1.e-7;
      double maxdelta = 0.;
      for (size_t ind = 0; ind < nptsOutput; ind++)
        {
          T_OUT outputSpiralPoint = outputSpiralHostPtr[ind];
          T_OUT outputDeviceFFTPoint = outputDeviceFFTHostPtr[ind];
          // auto diffPoint = outputSpiralPoint - outputDeviceFFTPoint;
          // double diffReal = outputSpiralPoint.x - outputDeviceFFTPoint.x;
          // double diffImag = outputSpiralPoint.y - outputDeviceFFTPoint.y;
          double diffAbsPoint = diffAbs(outputSpiralPoint, outputDeviceFFTPoint);
          updateMaxAbs(maxdelta, diffAbsPoint);
          bool correctPoint = (diffAbsPoint < tol);
          if (!correctPoint)
            {
              correct = false;
            }
        }
      printf( "Correct: %s\tMax delta = %E\t\t##PICKME## \n",
              (correct ? "True" : "False"), maxdelta );
    }
  else
    {
      printf( "Correct: Could not compare.\t\t##PICKME## \n" );
    }

  // FIXME: writeBufferToFile
  
  delete[] outputSpiralHostPtr;
  delete[] outputDeviceFFTHostPtr;

  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tFIRST iteration\t##PICKME## \n",
         spiral_gpu[0], deviceFFT_gpu[0]);
  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tSECOND iteration\t##PICKME## \n",
         spiral_gpu[1], deviceFFT_gpu[1]);

  float avgSpiral = avgSubarray(spiral_gpu, BASE_ITERS, iters-1);
  float avgDevice = avgSubarray(deviceFFT_gpu, BASE_ITERS, iters-1);
  delete[] spiral_gpu;
  delete[] deviceFFT_gpu;

  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft), AVERAGE over %d iterations (range: 11 - %d) ##PICKME## \n",
         avgSpiral, avgDevice, NUM_ITERS, BASE_ITERS + NUM_ITERS );
}


int main(int argc, char* argv[])
{
  int iloop = 0;
  bool oneshot = false;
  TransformType ttype;
  int M, N, K;

  printf("Usage:  %s mddft|imddft|mdprdft|imdprdft [iterations=20] [MxNxK] [writefiles]\n",
         argv[0]);

  if (argc > 1)
    {
      int libmode;
      if (std::string(argv[1]) == "mddft")
        {
          ttype = MDDFT;
          libmode = fftx_mddft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "imddft")
        {
          ttype = IMDDFT;
          libmode = fftx_imddft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "mdprdft")
        {
          ttype = MDPRDFT;
          libmode = fftx_mdprdft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "imdprdft")
        {
          ttype = IMDPRDFT;
          libmode = fftx_imdprdft_GetLibraryMode();
        }
      else
        {
          printf("%s: failed to specify one of mddft|imddft|mdprdft|imdprdft\n",
                 argv[0]);
          exit(-1);
        }
      if ( (libmode != LIB_MODE_CUDA ) &&
           (libmode != LIB_MODE_HIP ) )
        { // Test is to time on a GPU [CUDA or HIP];
          // check library support this mode
          printf ( "%s: fftx_%s library doesn't support GPU, exiting...\n",
                   argv[0], argv[1] );
          exit (-1);
        }

      if (argc > 2)
        {
          NUM_ITERS = atoi ( argv[2] );

          if (argc > 3)
            { // Run size specified in form MxNxK.
              char * foo = argv[3];
              M = atoi ( foo );
              while ( * foo != 'x' ) foo++;
              foo++ ;
              N = atoi ( foo );
              while ( * foo != 'x' ) foo++;
              foo++ ;
              K = atoi ( foo );
              oneshot = true;
              printf ( "Run size: %dx%dx%d, ", M, N, K );

              if (argc > 4)
                { // Only write files when a specified [single] size is used.
                  // Write data to files:
                  // spiral input data, spiral output data, rocFFT/cuFFT output.
                  writefiles = true;
                  printf("WRITE data files\n");
                }
              else
                {
                  printf("DO NOT write data files\n");
                }
            }
          else
            {
              printf ( "Run all sizes found in library, DO NOT write data files\n" );
            }
        }
    }
  else
    {
      printf("%s: failed to specify one of mddft|imddft|mdprdft|imdprdft",
             argv[0]);
      exit(-1);
    }

  int iters = NUM_ITERS + BASE_ITERS;
  printf ( "%s %s: Measure %d iterations\n",
           argv[0], argv[1], iters );

  fftx::point_t<3> *wcube, curr;

  // last entry is { 0, 0, 0 }
  if (ttype == MDDFT)
    {
      wcube = fftx_mddft_QuerySizes ();
    }
  else if (ttype == IMDDFT)
    {
      wcube = fftx_imddft_QuerySizes ();
    }
  else if (ttype == MDPRDFT)
    {
      wcube = fftx_mdprdft_QuerySizes ();
    }
  else if (ttype == IMDPRDFT)
    {
      wcube = fftx_imdprdft_QuerySizes ();
    }
  if (wcube == NULL)
    {
      printf ( "%s %s: Failed to get list of available sizes\n",
               argv[0], argv[1] );
      exit (-1);
    }

  if ( oneshot )
    {
      for ( iloop = 0; ; iloop++ )
        {
          if ( wcube[iloop].x[0] == 0 &&
               wcube[iloop].x[1] == 0 &&
               wcube[iloop].x[2] == 0 )
            {
              //  requested size is not in library, print message & exit
              printf ( "%s %s: Cube { %d, %d, %d } not found in library ... exiting\n",
                       argv[0], argv[1], M, N, K );
              exit (-1);
            }
          if ( wcube[iloop].x[0] == M &&
               wcube[iloop].x[1] == N &&
               wcube[iloop].x[2] == K )
            {
              break;
            }
        }
      // Now wcube[iloop] is set to [M, N, K].
    }

#if defined(FFTX_HIP)
  //  Set up the library.
  rocfft_setup();
#endif

  double *X, *Y;
  // double sym[100];  // dummy symbol
  // transformTuple_t *tupl;

  for ( /* iloop is initialized */ ; ; iloop++ )
    {
      curr = wcube[iloop];
      if ( curr[0] == 0 &&
           curr[1] == 0 &&
           curr[2] == 0 )
        { // This is the end.
          break;
        }

      // M = curr.x[0];
      // N = curr.x[1];
      // K = curr.x[2];
      printf ( "Cube size { %d, %d, %d } is available\n",
               curr[0], curr[1], curr[2] );
      // If tupl == NULL then compareSize function will catch it.
      if (ttype == MDDFT)
        {
          transformTuple_t *tupl = fftx_mddft_Tuple ( curr );
          compareSize(curr, tupl, mddftDevice);
        }
      else if (ttype == IMDDFT)
        {
          transformTuple_t *tupl = fftx_imddft_Tuple ( curr );
          compareSize(curr, tupl, imddftDevice);
        }
      else if (ttype == MDPRDFT)
        {
          transformTuple_t *tupl = fftx_mdprdft_Tuple ( curr );
          compareSize(curr, tupl, mdprdftDevice);
        }
      else if (ttype == IMDPRDFT)
        {
          transformTuple_t *tupl = fftx_imdprdft_Tuple ( curr );
          compareSize(curr, tupl, imdprdftDevice);
        }

      if (oneshot)
        {
          break;
        }
    }

#if defined(FFTX_HIP)
  //  cleanup the library
  rocfft_cleanup();
#endif
  
  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
