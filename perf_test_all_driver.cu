#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fftxdevice_macros.h"

#include "fftx_mddft_public.h"
#include "fftx_imddft_public.h"
#include "fftx_mdprdft_public.h"
#include "fftx_imdprdft_public.h"
// #include "fftx_rconv_public.h"

#include "fftx3utilities.h"

enum TransformType { MDDFT, IMDDFT, MDPRDFT, IMDPRDFT };

static bool writefiles = false;

template<typename T>
T avgSubarray(const T* arr, int lo, int hi)
{
  T tot = 0.;
  int len = 0;
  for (int i = lo; i <= hi; i++)
    {
      tot += arr[i];
      len++;
    }
  T avg = tot / (len * 1.);
  return avg;
}

void setRand(double& a_val)
{
  a_val = 1. - ((double) rand()) / (double) (RAND_MAX/2);
}

void setRand(std::complex<double>& a_val)
{
  double x, y;
  setRand(x);
  setRand(y);
  a_val = std::complex<double>(x, y);
}

double diffAbs(double a_x,
               double a_y)
{
  double diffNorm = a_x - a_y;
  if (diffNorm < 0.) diffNorm = -diffNorm;
  return diffNorm;
}

double diffAbs(std::complex<double>& a_x,
               std::complex<double>& a_y)
{
  double diffNorm = std::abs(a_x - a_y);
  return diffNorm;
}

FFTX_DEVICE_FFT_RESULT deviceExecD2Z(FFTX_DEVICE_FFT_HANDLE a_plan,
                                double* a_in,
                                std::complex<double>* a_out)
{
  return FFTX_DEVICE_FFT_EXECD2Z(a_plan,
                            (FFTX_DEVICE_FFT_DOUBLEREAL*) a_in,
                            (FFTX_DEVICE_FFT_DOUBLECOMPLEX*) a_out);
}


template<typename T_IN, typename T_OUT>
struct deviceTransform
{
  deviceTransform(FFTX_DEVICE_FFT_TYPE a_tp,
                  int a_dir = 0)
  {
    m_tp = a_tp;
    m_dir = a_dir;
  }
                  
  FFTX_DEVICE_FFT_TYPE m_tp;

  int m_dir;

  FFTX_DEVICE_FFT_RESULT plan3d(FFTX_DEVICE_FFT_HANDLE& a_plan,
                           fftx::point_t<3> a_tfmSize)
  {
    return FFTX_DEVICE_FFT_PLAN3D(&a_plan,
                             a_tfmSize[0], a_tfmSize[1], a_tfmSize[2],
                             m_tp);
  }

  FFTX_DEVICE_FFT_RESULT exec(FFTX_DEVICE_FFT_HANDLE a_plan,
                         T_IN* a_in,
                         T_OUT* a_out)
  {
    if (m_tp == FFTX_DEVICE_FFT_Z2Z)
      {
        return FFTX_DEVICE_FFT_EXECZ2Z(a_plan,
                                  (FFTX_DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (FFTX_DEVICE_FFT_DOUBLECOMPLEX*) a_out,
                                  m_dir);
      }
    else if (m_tp == FFTX_DEVICE_FFT_D2Z)
      {
        return FFTX_DEVICE_FFT_EXECD2Z(a_plan,
                                  (FFTX_DEVICE_FFT_DOUBLEREAL*) a_in,
                                  (FFTX_DEVICE_FFT_DOUBLECOMPLEX*) a_out);
      }
    else if (m_tp == FFTX_DEVICE_FFT_Z2D)
      {
        return FFTX_DEVICE_FFT_EXECZ2D(a_plan,
                                  (FFTX_DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (FFTX_DEVICE_FFT_DOUBLEREAL*) a_out);
      }
    else
      {
        return (FFTX_DEVICE_FFT_RESULT) -1;
      }
  }
};
  

deviceTransform<std::complex<double>, std::complex<double> >
mddftDevice(FFTX_DEVICE_FFT_Z2Z, FFTX_DEVICE_FFT_FORWARD);

deviceTransform<std::complex<double>, std::complex<double> >
imddftDevice(FFTX_DEVICE_FFT_Z2Z, FFTX_DEVICE_FFT_INVERSE);

deviceTransform<double, std::complex<double> >
mdprdftDevice(FFTX_DEVICE_FFT_D2Z);

deviceTransform<std::complex<double>, double>
imdprdftDevice(FFTX_DEVICE_FFT_Z2D);

template<typename T_IN, typename T_OUT>
void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                T_IN* a_inPtr,
                T_OUT* a_outPtr);

void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                std::complex<double>* a_inPtr,
                std::complex<double>* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
}


void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                double* a_inPtr,
                std::complex<double>* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
  // Halve the domain of the complex array.
#if FFTX_COMPLEX_TRUNC_LAST
  a_outSize[2] = a_outSize[2]/2 + 1;
#else
  a_outSize[0] = a_outSize[0]/2 + 1;
#endif
}


void inoutSizes(fftx::point_t<3>& a_inSize,
                fftx::point_t<3>& a_outSize,
                const fftx::point_t<3>& a_fullSize,
                std::complex<double>* a_inPtr,
                double* a_outPtr)
{
  a_inSize = a_fullSize;
  a_outSize = a_fullSize;
  // Halve the domain of the complex array.
#if FFTX_COMPLEX_TRUNC_LAST
  a_inSize[2] = a_inSize[2]/2 + 1;
#else
  a_inSize[0] = a_inSize[0]/2 + 1;
#endif
}


static int NUM_ITERS = 100;
static int BASE_ITERS = 10;

template<typename T_IN, typename T_OUT>
void compareSize(fftx::point_t<3> a_size,
                 transformTuple_t *a_tupl,
                 deviceTransform<T_IN, T_OUT>& a_tfmDevice)
{
  bool doDevice = true;
  bool doSpiral = true;

  if (a_tupl == NULL)
    {
      doSpiral = false;
      printf ( "Failed to get tuple for cube { %d, %d, %d }\n",
               a_size[0], a_size[1], a_size[2]);
    }

  /*
    Allocate space for arrays, and set input array.
  */
  fftx::point_t<3> inputSize, outputSize;
  T_IN* inPtr;
  T_OUT* outPtr;
  inoutSizes(inputSize, outputSize, a_size, inPtr, outPtr);
  

  // This doesn't work. :/
  // const fftx::point_t<3> unit = fftx::point_t<3>::Unit();
  //   fftx::box_t<3> inputDomain(unit, inputSize);
  //   fftx::box_t<3> outputDomain(unit, outputSize);

  fftx::box_t<3> inputDomain(fftx::point_t<3>({{1, 1, 1}}),
                             fftx::point_t<3>({{inputSize[0],
                                                inputSize[1],
                                                inputSize[2]}}));
  fftx::box_t<3> outputDomain(fftx::point_t<3>({{1, 1, 1}}),
                              fftx::point_t<3>({{outputSize[0],
                                                 outputSize[1],
                                                 outputSize[2]}}));
  
  fftx::array_t<3, T_IN> inputArrayHost(inputDomain);
  size_t nptsInput = inputDomain.size();
  size_t nptsOutput = outputDomain.size();
  size_t bytesInput = nptsInput * sizeof(T_IN);
  size_t bytesOutput = nptsOutput * sizeof(T_OUT);
  forall([](T_IN(&v), const fftx::point_t<3>& p)
         {
           setRand(v);
         }, inputArrayHost);
  // This symmetrizes only for complex input and real output,
  // in order to get a complex array that transforms to a real array.
  fftx::array_t<3, T_OUT> outputArrayHost(outputDomain);
  symmetrizeHermitian(inputArrayHost, outputArrayHost);

  T_IN* inputHostPtr = inputArrayHost.m_data.local();
  // additional code for GPU programs
  T_IN* inputDevicePtr;
  T_OUT* outputSpiralDevicePtr;
  T_OUT* outputDeviceFFTDevicePtr;
  FFTX_DEVICE_MALLOC(&inputDevicePtr, bytesInput);
  FFTX_DEVICE_MALLOC(&outputSpiralDevicePtr, bytesOutput);
  FFTX_DEVICE_MALLOC(&outputDeviceFFTDevicePtr, bytesOutput);
  // Do this at the beginning of each iteration instead of here.
  //  FFTX_DEVICE_MEM_COPY(inputDevicePtr, inputHostPtr, // dest, source
  //                  npts*sizeof(double), // bytes
  //                  FFTX_MEM_COPY_HOST_TO_DEVICE); // type
  
  /*
    Set up timers for deviceFFT.
   */
  FFTX_DEVICE_EVENT_T spiralFFT_start, spiralFFT_stop;
  FFTX_DEVICE_EVENT_T deviceFFT_start, deviceFFT_stop;
  FFTX_DEVICE_EVENT_CREATE ( &spiralFFT_start );
  FFTX_DEVICE_EVENT_CREATE ( &spiralFFT_stop );
  FFTX_DEVICE_EVENT_CREATE ( &deviceFFT_start );
  FFTX_DEVICE_EVENT_CREATE ( &deviceFFT_stop );

  int iters = NUM_ITERS + BASE_ITERS;

  /*
    Get plan for deviceFFT.
  */
  // printf("get deviceFFT plan\n");
  FFTX_DEVICE_FFT_HANDLE plan;
  {
    auto rc = a_tfmDevice.plan3d(plan, a_size);
    if (rc != FFTX_DEVICE_FFT_SUCCESS)
      {
        printf ( "Create FFTX_DEVICE_FFT_PLAN3D failed with error code %d ... skip buffer check\n",
                 rc );
        doDevice = false;
      }
  }

  /*
    Time iterations of real-to-complex deviceFFT calls using the plan.
   */
  // printf("call deviceExec %d times\n", a_iterations);

  float* deviceFFT_gpu = new float[iters];
  for (int i = 0; i < iters; i++)
    {
      deviceFFT_gpu[i] = 0.;
    }
  if (doDevice)
    {
      for (int itn = 0; itn < iters; itn++ )
        {
          FFTX_DEVICE_MEM_COPY(inputDevicePtr, // dest
                          inputHostPtr, // source
                          bytesInput, // bytes
                          FFTX_MEM_COPY_HOST_TO_DEVICE); // type
          FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR() );
          FFTX_DEVICE_EVENT_RECORD( deviceFFT_start );
          int rc = a_tfmDevice.exec(plan,
                                    inputDevicePtr,
                                    outputDeviceFFTDevicePtr);
          if (rc != FFTX_DEVICE_FFT_SUCCESS)
            {
              printf ( "Launch device exec failed with error code %d ... skip buffer check\n",
                       rc );
              doDevice = false;
              break;
            }
          FFTX_DEVICE_EVENT_RECORD( deviceFFT_stop );
          FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR() );
          FFTX_DEVICE_EVENT_SYNCHRONIZE( deviceFFT_stop );
          FFTX_DEVICE_EVENT_ELAPSED_TIME( &deviceFFT_gpu[itn],
                                     deviceFFT_start,
                                     deviceFFT_stop );
        }
    }
  FFTX_DEVICE_FFT_DESTROY(plan);

  FFTX_DEVICE_SYNCHRONIZE();

  // printf("call Spiral transform %d times\n", iters);

  /*
    Time iterations of transform with SPIRAL-generated code.
   */
  float* spiral_gpu = new float[iters];
  for (int i = 0; i < iters; i++)
    {
      spiral_gpu[i] = 0.;
    }

  FFTX_DEVICE_MEM_COPY(inputDevicePtr, // dest
                  inputHostPtr, // source
                  bytesInput, // bytes
                  FFTX_MEM_COPY_HOST_TO_DEVICE); // type
  FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR() );

  if (doSpiral)
    {
      double sym[100];  // dummy symbol
      ( * a_tupl->initfp )();
      FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR () );

      for (int itn = 0; itn < iters; itn++)
        {
          FFTX_DEVICE_EVENT_RECORD( spiralFFT_start );
          ( * a_tupl->runfp ) ( (double*) outputSpiralDevicePtr,
                                (double*) inputDevicePtr,
                                sym );
          FFTX_DEVICE_EVENT_RECORD( spiralFFT_stop );
          FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR () );
          FFTX_DEVICE_EVENT_SYNCHRONIZE ( spiralFFT_stop );
          FFTX_DEVICE_EVENT_ELAPSED_TIME ( &spiral_gpu[itn],
                                      spiralFFT_start,
                                      spiralFFT_stop );
        }

      //  Call the destroy function
      ( * a_tupl->destroyfp )();
      FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR () );
    }
  
  /*
    Check that deviceFFT and SPIRAL give the same results on last iteration.
  */
  T_OUT* outputSpiralHostPtr = new T_OUT[nptsOutput];
  T_OUT* outputDeviceFFTHostPtr = new T_OUT[nptsOutput];
  FFTX_DEVICE_MEM_COPY(outputSpiralHostPtr, // dest
                  outputSpiralDevicePtr, // source
                  bytesOutput, // bytes
                  FFTX_MEM_COPY_DEVICE_TO_HOST); // type
  FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR() );
  FFTX_DEVICE_MEM_COPY(outputDeviceFFTHostPtr, // dest
                  outputDeviceFFTDevicePtr, // source
                  bytesOutput, // bytes
                  FFTX_MEM_COPY_DEVICE_TO_HOST); // type
  FFTX_DEVICE_CHECK_ERROR ( FFTX_DEVICE_GET_LAST_ERROR() );

  FFTX_DEVICE_FREE(inputDevicePtr);
  FFTX_DEVICE_FREE(outputSpiralDevicePtr);
  FFTX_DEVICE_FREE(outputDeviceFFTDevicePtr);

  printf("cube = [ %d, %d, %d ]\t", a_size[0], a_size[1], a_size[2]);
  if (doSpiral && doDevice)
    {
      bool correct = true;
      const double tol = 1.e-7;
      double maxdelta = 0.;
      for (size_t ind = 0; ind < nptsOutput; ind++)
        {
          T_OUT outputSpiralPoint = outputSpiralHostPtr[ind];
          T_OUT outputDeviceFFTPoint = outputDeviceFFTHostPtr[ind];
          // auto diffPoint = outputSpiralPoint - outputDeviceFFTPoint;
          // double diffReal = outputSpiralPoint.x - outputDeviceFFTPoint.x;
          // double diffImag = outputSpiralPoint.y - outputDeviceFFTPoint.y;
          double diffAbsPoint = diffAbs(outputSpiralPoint, outputDeviceFFTPoint);
          updateMaxAbs(maxdelta, diffAbsPoint);
          bool correctPoint = (diffAbsPoint < tol);
          if (!correctPoint)
            {
              correct = false;
            }
        }
      printf( "Correct: %s\tMax delta = %E\t\t##PICKME## \n",
              (correct ? "True" : "False"), maxdelta );
    }
  else
    {
      printf( "Correct: Could not compare.\t\t##PICKME## \n" );
    }

  // FIXME: writeBufferToFile
  
  delete[] outputSpiralHostPtr;
  delete[] outputDeviceFFTHostPtr;

  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tFIRST iteration\t##PICKME## \n",
         spiral_gpu[0], deviceFFT_gpu[0]);
  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft),\t\tSECOND iteration\t##PICKME## \n",
         spiral_gpu[1], deviceFFT_gpu[1]);

  float avgSpiral = avgSubarray(spiral_gpu, BASE_ITERS, iters-1);
  float avgDevice = avgSubarray(deviceFFT_gpu, BASE_ITERS, iters-1);
  delete[] spiral_gpu;
  delete[] deviceFFT_gpu;

  printf("%f\tms (SPIRAL) vs\t%f\tms (hipfft), AVERAGE over %d iterations (range: 11 - %d) ##PICKME## \n",
         avgSpiral, avgDevice, NUM_ITERS, BASE_ITERS + NUM_ITERS );
}


int main(int argc, char* argv[])
{
  int iloop = 0;
  bool oneshot = false;
  TransformType ttype;
  int M, N, K;

  printf("Usage:  %s mddft|imddft|mdprdft|imdprdft [iterations=20] [MxNxK] [writefiles]\n",
         argv[0]);

  if (argc > 1)
    {
      int libmode;
      if (std::string(argv[1]) == "mddft")
        {
          ttype = MDDFT;
          libmode = fftx_mddft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "imddft")
        {
          ttype = IMDDFT;
          libmode = fftx_imddft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "mdprdft")
        {
          ttype = MDPRDFT;
          libmode = fftx_mdprdft_GetLibraryMode();
        }
      else if (std::string(argv[1]) == "imdprdft")
        {
          ttype = IMDPRDFT;
          libmode = fftx_imdprdft_GetLibraryMode();
        }
      else
        {
          printf("%s: failed to specify one of mddft|imddft|mdprdft|imdprdft\n",
                 argv[0]);
          exit(-1);
        }
      if ( (libmode != LIB_MODE_CUDA ) &&
           (libmode != LIB_MODE_HIP ) )
        { // Test is to time on a GPU [CUDA or HIP];
          // check library support this mode
          printf ( "%s: fftx_%s library doesn't support GPU, exiting...\n",
                   argv[0], argv[1] );
          exit (-1);
        }

      if (argc > 2)
        {
          NUM_ITERS = atoi ( argv[2] );

          if (argc > 3)
            { // Run size specified in form MxNxK.
              char * foo = argv[3];
              M = atoi ( foo );
              while ( * foo != 'x' ) foo++;
              foo++ ;
              N = atoi ( foo );
              while ( * foo != 'x' ) foo++;
              foo++ ;
              K = atoi ( foo );
              oneshot = true;
              printf ( "Run size: %dx%dx%d, ", M, N, K );

              if (argc > 4)
                { // Only write files when a specified [single] size is used.
                  // Write data to files:
                  // spiral input data, spiral output data, rocFFT/cuFFT output.
                  writefiles = true;
                  printf("WRITE data files\n");
                }
              else
                {
                  printf("DO NOT write data files\n");
                }
            }
          else
            {
              printf ( "Run all sizes found in library, DO NOT write data files\n" );
            }
        }
    }
  else
    {
      printf("%s: failed to specify one of mddft|imddft|mdprdft|imdprdft",
             argv[0]);
      exit(-1);
    }

  int iters = NUM_ITERS + BASE_ITERS;
  printf ( "%s %s: Measure %d iterations\n",
           argv[0], argv[1], iters );

  fftx::point_t<3> *wcube, curr;

  // last entry is { 0, 0, 0 }
  if (ttype == MDDFT)
    {
      wcube = fftx_mddft_QuerySizes ();
    }
  else if (ttype == IMDDFT)
    {
      wcube = fftx_imddft_QuerySizes ();
    }
  else if (ttype == MDPRDFT)
    {
      wcube = fftx_mdprdft_QuerySizes ();
    }
  else if (ttype == IMDPRDFT)
    {
      wcube = fftx_imdprdft_QuerySizes ();
    }
  if (wcube == NULL)
    {
      printf ( "%s %s: Failed to get list of available sizes\n",
               argv[0], argv[1] );
      exit (-1);
    }

  if ( oneshot )
    {
      for ( iloop = 0; ; iloop++ )
        {
          if ( wcube[iloop].x[0] == 0 &&
               wcube[iloop].x[1] == 0 &&
               wcube[iloop].x[2] == 0 )
            {
              //  requested size is not in library, print message & exit
              printf ( "%s %s: Cube { %d, %d, %d } not found in library ... exiting\n",
                       argv[0], argv[1], M, N, K );
              exit (-1);
            }
          if ( wcube[iloop].x[0] == M &&
               wcube[iloop].x[1] == N &&
               wcube[iloop].x[2] == K )
            {
              break;
            }
        }
      // Now wcube[iloop] is set to [M, N, K].
    }

#if defined(FFTX_HIP)
  //  Set up the library.
  rocfft_setup();
#endif

  double *X, *Y;
  // double sym[100];  // dummy symbol
  // transformTuple_t *tupl;

  for ( /* iloop is initialized */ ; ; iloop++ )
    {
      curr = wcube[iloop];
      if ( curr[0] == 0 &&
           curr[1] == 0 &&
           curr[2] == 0 )
        { // This is the end.
          break;
        }

      // M = curr.x[0];
      // N = curr.x[1];
      // K = curr.x[2];
      printf ( "Cube size { %d, %d, %d } is available\n",
               curr[0], curr[1], curr[2] );
      // If tupl == NULL then compareSize function will catch it.
      if (ttype == MDDFT)
        {
          transformTuple_t *tupl = fftx_mddft_Tuple ( curr );
          compareSize(curr, tupl, mddftDevice);
        }
      else if (ttype == IMDDFT)
        {
          transformTuple_t *tupl = fftx_imddft_Tuple ( curr );
          compareSize(curr, tupl, imddftDevice);
        }
      else if (ttype == MDPRDFT)
        {
          transformTuple_t *tupl = fftx_mdprdft_Tuple ( curr );
          compareSize(curr, tupl, mdprdftDevice);
        }
      else if (ttype == IMDPRDFT)
        {
          transformTuple_t *tupl = fftx_imdprdft_Tuple ( curr );
          compareSize(curr, tupl, imdprdftDevice);
        }

      if (oneshot)
        {
          break;
        }
    }

#if defined(FFTX_HIP)
  //  cleanup the library
  rocfft_cleanup();
#endif
  
  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
